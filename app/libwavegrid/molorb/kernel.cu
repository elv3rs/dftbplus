#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------------------------------*
 *  DFTB+: general package for performing fast atomistic simulations                               *
 *  Copyright (C) 2006 - 2025  DFTB+ developers group                                              *
 *                                                                                                 *
 *  See the LICENSE file for terms of usage and distribution.                                      *
 *-------------------------------------------------------------------------------------------------*/
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <omp.h>
#include <cstdio>
#include <cmath>
#include <algorithm>
#include <assert.h>

#include "kernel.cuh"
#include "utils.cuh"
#include "slater.cuh"

// more print statements
constexpr bool debug = true; 
constexpr double INV_R_EPSILON = 1.0e-12;
// amount of shared memory set aside for nEig accumulators
constexpr float SHARED_MEM_FACTOR = 0.95f;
// max output array share of free global memory
constexpr float GLOBAL_MEM_FACTOR = 0.80f;
// Threads per block, multiple of warp size 32
constexpr int block_size = 256; 

using complexd = thrust::complex<double>;

// Manages Gpu memory allocation
struct DeviceData {
    // Grid
    DeviceBuffer<double> origin;
    DeviceBuffer<double> gridVecs;

    // System
    DeviceBuffer<double> coords;
    DeviceBuffer<int>    species;
    DeviceBuffer<int>    iStos;

    // Periodic
    DeviceBuffer<double> latVecs;
    DeviceBuffer<double> recVecs2pi;
    DeviceBuffer<int>    kIndexes;
    DeviceBuffer<complexd> phases;

    // STO Basis
    DeviceBuffer<int>    sto_angMoms;
    DeviceBuffer<int>    sto_nPows;
    DeviceBuffer<int>    sto_nAlphas;
    DeviceBuffer<double> sto_cutoffsSq;
    DeviceBuffer<double> sto_coeffs;
    DeviceBuffer<double> sto_alphas;
    // Texture for radial LUT
    hipTextureObject_t sto_lutTex;

    // Eigenvectors
    DeviceBuffer<double> eigVecsReal;
    DeviceBuffer<complexd> eigVecsCmpl;

    // Constructor handles all H2D allocation and copy
    DeviceData(const GridParams* grid, const SystemParams* system, const PeriodicParams* periodic, const StoBasisParams* basis, const CalculationParams* calc)
        : origin(grid->origin, 3),
          gridVecs(grid->gridVecs, 9),
          coords(system->coords, (size_t)3 * system->nAtom * system->nCell),
          species(system->species, system->nAtom),
          iStos(system->iStos, system->nSpecies + 1),
    {
        if (basis->useRadialLut) {
            // Convert the Fortran passed doubles to floats
            size_t totalLutValues = (size_t)basis->nStos * basis->nLutPoints;
            std::vector<float> lutGridValuesFloat(totalLutValues);
            for (size_t i = 0; i < totalLutValues; ++i) {
                lutGridValuesFloat[i] = (float)(basis->lutGridValues[i]);
            }

            // Allocate
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
            hipExtent shape = make_hipExtent(basis->nLutPoints, basis->nStos, 0);
            hipArray_t lutArray;
            CHECK_CUDA(hipMalloc3DArray(&lutArray, &channelDesc, shape, hipArrayLayered));

            // Copy data to array
            hipMemcpy2DToArray(
                lutArray,                              // dst array
                0, 0,                                  // no offset in dst
                lutGridValuesFloat.data(),             // src pointer
                basis->nLutPoints * sizeof(float),     // src pitch (for alignment, bytes tp next row)
                basis->nLutPoints * sizeof(float),     // width in bytes
                basis->nStos,                          // height (number of cached stos)
                hipMemcpyHostToDevice
            );

            // Prepare texture object properties
            hipResourceDesc resDesc{};
            resDesc.resType = hipResourceTypeArray;
            resDesc.res.array.array = lutArray;

            hipTextureDesc texDesc{};
            texDesc.addressMode[0] = hipAddressModeClamp;
            texDesc.addressMode[1] = hipAddressModeClamp;
            texDesc.filterMode = hipFilterModeLinear; 
            texDesc.readMode = hipReadModeElementType; // dont normalize our lut values
            texDesc.normalizedCoords = 0; // access using [0, N-1]
            // Create texture object
            CHECK_CUDA(hipCreateTextureObject(&sto_lutTex, &resDesc, &texDesc, nullptr));

        } else {
            sto_angMoms.assign(basis->sto_angMoms, basis->nStos);
            sto_nPows.assign(basis->sto_nPows, basis->nStos);
            sto_nAlphas.assign(basis->sto_nAlphas, basis->nStos);
            sto_cutoffsSq.assign(basis->sto_cutoffsSq, basis->nStos);
            sto_coeffs.assign(basis->sto_coeffs, (size_t)basis->maxNPows * basis->maxNAlphas * basis->nStos);
            sto_alphas.assign(basis->sto_alphas, (size_t)basis->maxNAlphas * basis->nStos);
        }

        if (calc->isRealInput) {
            eigVecsReal.assign(calc->eigVecsReal, (size_t)system->nOrb * calc->nEigIn);
        } else {
            eigVecsCmpl.assign(reinterpret_cast<const complexd*>(calc->eigVecsCmpl), (size_t)system->nOrb * calc->nEigIn);
            phases.assign(reinterpret_cast<const complexd*>(periodic->phases), (size_t)system->nCell * calc->nEigIn);
            kIndexes.assign(periodic->kIndexes, calc->nEigIn);
        }
        if (periodic->isPeriodic) {
            latVecs.assign(periodic->latVecs, 9);
            recVecs2pi.assign(periodic->recVecs2pi, 9);
        }
    }
};

// Kernel parameters
struct DeviceKernelParams {
    // Grid
    int nPointsX, nPointsY, nPointsZ_batch, z_offset_global;
    const double* origin;
    const double* gridVecs;

    // System
    int nAtom, nCell, nOrb;
    const double* coords;
    const int*    species;
    const int*    iStos;

    // Periodic boundary cond.
    bool isPeriodic;
    const double* latVecs;
    const double* recVecs2pi;
    const int*    kIndexes;
    const complexd* phases;

    // STO Basis
    int nStos, maxNPows, maxNAlphas;
    // Texture LUTs
    cudaTextureObject* lutTex;
    double inverseLutStep;
    // STO parameters
    const int*    sto_angMoms;
    const int*    sto_nPows;
    const int*    sto_nAlphas;
    const double* sto_cutoffsSq;
    const double* sto_coeffs;
    const double* sto_alphas;

    // Eigenvectors
    int nEig, nEig_per_pass;
    const double* eigVecsReal;
    const complexd* eigVecsCmpl;

    // Output (batch pointers), varied in batch loop
    double* valueReal_out_batch;
    complexd* valueCmpl_out_batch;

    // Constructor to initialize the parameters from host data
    // Batch-specific parameters are initialized to zero or nullptr,
    // and need to be set in the loop before kernel launch.
    DeviceKernelParams(
        const DeviceData& data,
        const GridParams* grid,
        const SystemParams* system,
        const PeriodicParams* periodic,
        const StoBasisParams* basis,
        const CalculationParams* calc
    ) {
        // Grid
        origin = data.origin.get();
        gridVecs = data.gridVecs.get();
        nPointsX = grid->nPointsX;
        nPointsY = grid->nPointsY;

        // System
        nAtom = system->nAtom;
        nCell = system->nCell;
        nOrb = system->nOrb;
        coords = data.coords.get();
        species = data.species.get();
        iStos = data.iStos.get();

        // STO Basis
        nStos = basis->nStos;
        if (basis->useRadialLut) {
            lutTex = data.sto_lutTex;
            inverseLutStep = basis->inverseLutStep;
        } else {
            maxNPows = basis->maxNPows;
            maxNAlphas = basis->maxNAlphas;
            sto_angMoms = data.sto_angMoms.get();
            sto_nPows = data.sto_nPows.get();
            sto_nAlphas = data.sto_nAlphas.get();
            sto_cutoffsSq = data.sto_cutoffsSq.get();
            sto_coeffs = data.sto_coeffs.get();
            sto_alphas = data.sto_alphas.get();
        }


        // Periodic boundary conditions
        isPeriodic = periodic->isPeriodic;
        latVecs = data.latVecs.get();
        recVecs2pi = data.recVecs2pi.get();
        kIndexes = data.kIndexes.get();
        phases = data.phases.get();

        // Eigenvectors
        nEig = calc->nEigIn;
        eigVecsReal = data.eigVecsReal.get();
        eigVecsCmpl = data.eigVecsCmpl.get();

        // Batch-specific kernel config to be updated in the loop
        nPointsZ_batch = 0;
        z_offset_global = 0;
        nEig_per_pass = 0;
        valueReal_out_batch = nullptr;
        valueCmpl_out_batch = nullptr;
    }
};



// =========================================================================
//  CUDA Kernel.
// =========================================================================
// To avoid branching (dropped at compile time), we template the kernel 16 ways on (isRealInput, calcDensity, calcTotalChrg, useLUT).
// useLUT decides whether to use texture memory interpolation for STO radial functions.
// isPeriodic decides whether to fold coords into unit cell.
// isRealInput decides whether to use real/complex eigenvectors (and adds phases)
// calcAtomicDensity squares the basis wavefunction contributions, result in valueReal_out of shape (x,y,z,n)
// calcTotalChrg accumulates the density over all states, leading to valueReal_out of shape (x,y,z,1).
// User is responsible for providing eigenvec multiplied with sqrt(occupation) if needed.
template <bool isRealInput, bool calcAtomicDensity, bool calcTotalChrg, bool useLUT>
__global__ void evaluateKernel(const DeviceKernelParams p)
{
    using AccumT = typename std::conditional<(isRealInput), double, complexd>::type;
    
    // Each thread gets its own private slice of the shared memory buffer for fast accumulation.
    // We have to chunk the eigenstates into nEig_per_pass due to size constraints.
    // (Cuda doesnt allow templating the shared memory type, so we simply recast it.)
    extern __shared__ char shared_workspace[];
    AccumT* point_results_pass = reinterpret_cast<AccumT*>(shared_workspace) + threadIdx.x * p.nEig_per_pass;


    // --- Thread to point mapping ---
    // Map each thread to unique 1d index
    int idx_in_batch = blockIdx.x * blockDim.x + threadIdx.x;
    int total_points_in_batch = p.nPointsX * p.nPointsY * p.nPointsZ_batch;
    if (idx_in_batch >= total_points_in_batch) return;

    // Map 1d index to point in grid
    int i1 = idx_in_batch % p.nPointsX;
    int i2 = (idx_in_batch / p.nPointsX) % p.nPointsY;
    int i3_batch = idx_in_batch / (p.nPointsX * p.nPointsY);
    int i3_global = i3_batch + p.z_offset_global; 

    // Map point to global coordinates.
    double xyz[3];
    for (int i = 0; i < 3; ++i) 
        xyz[i] = p.origin[i] + i1 * p.gridVecs[IDX2F(i, 0, 3)]
                             + i2 * p.gridVecs[IDX2F(i, 1, 3)]
                             + i3_global * p.gridVecs[IDX2F(i, 2, 3)];
    
    // If periodic, fold into cell by discarding the non-fractional part in lattice vector multiples.
    if (p.isPeriodic) 
        foldCoordsIntoCell(xyz, reinterpret_cast<const double (*)[3]>(p.latVecs), reinterpret_cast<const double (*)[3]>(p.recVecs2pi));
    

    double totChrgAcc = 0.0;
    // --- Loop over eigenstates in chunks that fit in shared memory ---
    for (int eig_base = 0; eig_base < p.nEig; eig_base += p.nEig_per_pass) {
        
        // Initialize the small, per-pass buffer for this thread
        for (int i = 0; i < p.nEig_per_pass; ++i) {
            point_results_pass[i] = AccumT(0.0);
        }

        // Since we run out of space in point_result_pass[], the spatial calculation 
        // is repeated for each chunk of eigenstates.
        // This is to keep the accumulation in fast shared memory.
        for (int iCell = 0; iCell < p.nCell; ++iCell) {
            int orbital_idx_counter = 0; 
            for (int iAtom = 0; iAtom < p.nAtom; ++iAtom) {
                int iSpecies = p.species[iAtom] - 1;
                double diff[3];
                for (int i = 0; i < 3; ++i) {
                    diff[i] = xyz[i] - p.coords[IDX3F(i, iAtom, iCell, 3, p.nAtom)];
                }
                double rr = diff[0] * diff[0] + diff[1] * diff[1] + diff[2] * diff[2];

                for (int iOrb = p.iStos[iSpecies] - 1; iOrb < p.iStos[iSpecies + 1] - 1; ++iOrb) {
                    int iL = p.sto_angMoms[iOrb];
                    if (rr > p.sto_cutoffsSq[iOrb]) {
                        orbital_idx_counter += 2 * iL + 1;
                        continue;
                    }
                    double r = sqrt(rr);

                    if constexpr (useLUT) {
                        double lut_pos = 0.5f + r * p.inverseLutStep;
                        double radialVal = static_cast<double>(tex1DLayered<float>(p.lutTex, lut_pos, iOrb)[0]);
                    } else {
                        double radialVal = getRadialValue(
                            r, iL, iOrb, p.sto_nPows[iOrb], p.sto_nAlphas[iOrb],
                            p.sto_coeffs, p.sto_alphas, p.maxNPows, p.maxNAlphas);
                    } 


                    // precompute inverse used across several realTessY calls
                    double inv_r = (r < INV_R_EPSILON) ? 0.0 : 1.0 / r;
                    double inv_r2 = inv_r * inv_r;

                    for (int iM = -iL; iM <= iL; ++iM) {
                        double val = radialVal * realTessY(iL, iM, diff, inv_r, inv_r2);
                        
                        // Accumulate into the small shared memory buffer for the current chunk
                        for (int iEig_offset = 0; iEig_offset < p.nEig_per_pass; ++iEig_offset) {
                            int iEig = eig_base + iEig_offset;
                            if (iEig >= p.nEig) break; // Don't go past the end on the last chunk
                            size_t eig_idx = IDX2F(orbital_idx_counter, iEig, p.nOrb);
                            if constexpr (isRealInput) {
                                point_results_pass[iEig_offset] += val * p.eigVecsReal[eig_idx];
                            } else {
                                point_results_pass[iEig_offset] += val * p.phases[IDX2F(iCell, iEig, p.nCell)] * p.eigVecsCmpl[eig_idx];
                            }
                        }
                        orbital_idx_counter++;
                    }
                }
            }
        }

        // Write the complete nEig_per_pass chunk to global memory.
        for (int iEig_offset = 0; iEig_offset < p.nEig_per_pass; ++iEig_offset) {
            int iEig = eig_base + iEig_offset;
            if (iEig >= p.nEig) break;
            size_t out_idx = IDX4F(i1, i2, i3_batch, iEig, p.nPointsX, p.nPointsY, p.nPointsZ_batch);
            if constexpr (isRealInput) {
                if constexpr (calcTotalChrg) {
                    totChrgAcc += point_results_pass[iEig_offset] * point_results_pass[iEig_offset];
                } else if (calcAtomicDensity) {
                    p.valueReal_out_batch[out_idx] = point_results_pass[iEig_offset] * point_results_pass[iEig_offset];
                } else {
                    p.valueReal_out_batch[out_idx] = point_results_pass[iEig_offset];
                }
            } else {
                if constexpr (calcTotalChrg) {
                    totChrgAcc += thrust::norm(point_results_pass[iEig_offset]);
                } else if constexpr (calcAtomicDensity) {
                    p.valueReal_out_batch[out_idx] = thrust::norm(point_results_pass[iEig_offset]);
                } else {
                    p.valueCmpl_out_batch[out_idx] = point_results_pass[iEig_offset];
                }
            }
        }
    }

    // Density stored in first eig : (x,y,z, 1)
    if constexpr (calcTotalChrg) {
        size_t out_idx = IDX4F(i1, i2, i3_batch, 0, p.nPointsX, p.nPointsY, p.nPointsZ_batch);
        p.valueReal_out_batch[out_idx] = totChrgAcc; 
    }
}



// =========================================================================
//  C++ Host Interface (callable from C/Fortran)
// =========================================================================
extern "C" void evaluate_on_device_c(
    const GridParams* grid,
    const SystemParams* system,
    const PeriodicParams* periodic,
    const StoBasisParams* basis,
    const CalculationParams* calc
){
    // Since we use these often, derefence them and add to namespace
    int nPointsX = grid->nPointsX;
    int nPointsY = grid->nPointsY;
    int nPointsZ = grid->nPointsZ;
    bool isRealOutput = calc->isRealInput || calc->calcAtomicDensity;

    if (calc->nEigIn == 0 || nPointsZ == 0) return; // Nothing to do
    if (calc->calcTotalChrg) {
        assert(calc->nEigOut == 1);
    } else {
        assert(calc->nEigOut == calc->nEigIn);
    }
    
    
    // We currently assume a hardcoded maximum for the number of powers.
    if (basis->maxNPows > STO_MAX_POWS) {
        fprintf(stderr, "Error: maxNPows (%d) exceeds STO_MAX_POWS (%d)\n", basis->maxNPows, STO_MAX_POWS);
        exit(EXIT_FAILURE);
    }
    
    // Timing events.
    hipEvent_t startEverything, endEverything;
    hipEvent_t startKernelOnly, endKernelOnly, startCopyOnly, endCopyOnly;
    float totalKernelTime_ms = 0.0f;
    float totalD2HCopyTime_ms = 0.0f;
    CHECK_CUDA(hipEventCreate(&startEverything));
    CHECK_CUDA(hipEventCreate(&endEverything));
    CHECK_CUDA(hipEventCreate(&startKernelOnly));
    CHECK_CUDA(hipEventCreate(&endKernelOnly));
    CHECK_CUDA(hipEventCreate(&startCopyOnly));
    CHECK_CUDA(hipEventCreate(&endCopyOnly));
    CHECK_CUDA(hipEventRecord(startEverything));


    // --- Multi-GPU Setup ---
    int numGpus;
    CHECK_CUDA(hipGetDeviceCount(&numGpus));
    if (numGpus == 0) {
        fprintf(stderr, "No CUDA-enabled GPUs found. Unable to launch Kernel.\n");
        exit(EXIT_FAILURE);
    }
    printf("Found %d GPUs.", numGpus);

#ifndef _OPENMP
    if (numGpus > 1) {
    printf("\nWARNING: Code not compiled with OpenMP support (-fopenmp). Falling back to single-GPU mode.\n");
    numGpus = 1;
    printf("Running on GPU 0 only.\n");
    }
#endif

    // Use OMP to split across available GPUs
    // This works irrespective of the number of threads set in OMP_NUM_THREADS.
    #pragma omp parallel num_threads(numGpus) 
    {
        int deviceId = omp_get_thread_num();
        CHECK_CUDA(hipSetDevice(deviceId));

        // --- Work Distribution: Divide Z-slices among GPUs ---
        int z_slices_per_gpu = nPointsZ / numGpus;
        int z_start_for_device = deviceId * z_slices_per_gpu;
        // Handle uneven Z-slice count
        int z_count_for_device = (deviceId == numGpus - 1) ? (nPointsZ - z_start_for_device) : z_slices_per_gpu;

        if (z_count_for_device > 0) {
            // --- Allocate on and copy data to gpu---
            DeviceData device_data(grid, system, periodic, basis, calc);

            // --- Per-GPU Kernel Configuration ---
            hipDeviceProp_t prop;
            CHECK_CUDA(hipGetDeviceProperties(&prop, deviceId));
            
            // Determine available shared memory for nEig_per_pass
            size_t available_shared = prop.sharedMemPerBlock * SHARED_MEM_FACTOR;
            size_t number_size = isRealOutput ? sizeof(double) : sizeof(complexd);
            int nEig_per_pass = available_shared / (block_size * number_size);
            if (nEig_per_pass == 0) nEig_per_pass = 1;
            if (nEig_per_pass > calc->nEigIn) nEig_per_pass = calc->nEigIn;
            size_t shared_mem_for_pass = (size_t)nEig_per_pass * block_size * number_size;

            
            // Determine the number of Z-slices to process in a single batch
            size_t free_mem, total_mem;
            CHECK_CUDA(hipMemGetInfo(&free_mem, &total_mem));
            size_t available_for_batch = static_cast<size_t>(free_mem * GLOBAL_MEM_FACTOR);
            size_t z_slice_size_bytes = (size_t)nPointsX * nPointsY * calc->nEigOut * number_size;
            
            // Determine max Z-slices that can fit in available (global) memory
            int z_batch_size = z_count_for_device; 
            if (z_slice_size_bytes > 0 && ((size_t)z_count_for_device * z_slice_size_bytes) > available_for_batch) {
                z_batch_size = available_for_batch / z_slice_size_bytes;
                if (z_batch_size == 0) z_batch_size = 1;
            }


            // Per-GPU batch buffer for the output
            size_t batch_buffer_size_elems = (size_t)nPointsX * nPointsY * std::min(z_count_for_device, z_batch_size) * calc->nEigOut;
            // todo: move this to deviceParams
            DeviceBuffer<complexd> d_valueCmpl_out_batch;
            DeviceBuffer<double> d_valueReal_out_batch;
            if (calc->isRealInput || calc->calcAtomicDensity) {
                d_valueReal_out_batch = DeviceBuffer<double>(batch_buffer_size_elems);
            } else {
                d_valueCmpl_out_batch = DeviceBuffer<complexd>(batch_buffer_size_elems);
            }

            // Debug output
            #pragma omp critical
            if (deviceId == 0 && debug) {
                printf("\n--- GPU %d (Lead) Configuration ---\n", deviceId);
                printf("  Z-slice workload: %d (from index %d to %d)\n", z_count_for_device, z_start_for_device, z_start_for_device + z_count_for_device - 1);
                printf("  Block size: %d threads, %zub shared mem per block, %d eigs per pass\n",
                    block_size, shared_mem_for_pass, nEig_per_pass);
                size_t total_size_valueOut = (size_t)nPointsX * nPointsY * nPointsZ * calc->nEigOut * sizeof(double);
                if (!calc->isRealInput && !calc->calcAtomicDensity) total_size_valueOut *= 2; 
                printf(" (Free device mem: %.2f GB, Grid size: %d x %d x %d (x %d eigs) = %.2f GB)\n",
                    free_mem / 1e9, nPointsX, nPointsY, nPointsZ, calc->nEigOut,
                    total_size_valueOut / 1e9);
                printf("  Processing Z-slices in batches of %d\n", z_batch_size);

            }

            // --- Populate Kernel Parameter struct ---
            DeviceKernelParams deviceParams(device_data, grid, system, periodic, basis, calc);
            deviceParams.nEig_per_pass = nEig_per_pass; // Set remaining params
            deviceParams.valueReal_out_batch = d_valueReal_out_batch.get();
            deviceParams.valueCmpl_out_batch = d_valueCmpl_out_batch.get();


            // --- Per-GPU Kernel Execution Loop ---
            // This loop iterates over the Z-slices assigned to *this* GPU.
            for (int z_offset_in_device_chunk = 0; z_offset_in_device_chunk < z_count_for_device; z_offset_in_device_chunk += z_batch_size) {
                deviceParams.nPointsZ_batch = std::min(z_batch_size, z_count_for_device - z_offset_in_device_chunk);

                int total_points_in_batch = nPointsX * nPointsY * deviceParams.nPointsZ_batch;
                if (total_points_in_batch == 0) continue;

                // The global z_offset is what the kernel needs to calculate correct coordinates
                deviceParams.z_offset_global = z_start_for_device + z_offset_in_device_chunk;
                int grid_size = (total_points_in_batch + block_size - 1) / block_size;
                if(deviceId == 0) {
                    CHECK_CUDA(hipEventRecord(startKernelOnly));
                }
                 
                #define CALL_KERNEL(isReal, doAtomic, doChrg, useLut) \
                    evaluateKernel<isReal, doAtomic, doChrg, useLut> \
                        <<<grid_size, block_size, shared_mem_for_pass>>>(deviceParams);

                int idx = (calc->isRealInput     ? 1 : 0)
                        + (calc->calcAtomicDensity ? 2 : 0)
                        + (calc->calcTotalChrg     ? 4 : 0);
                        + (basis->useRadialLut     ? 8 : 0);

                switch (idx) {
                    case 0:  CALL_KERNEL(false, false, false, false); break;
                    case 1:  CALL_KERNEL(true,  false, false, false); break;
                    case 2:  CALL_KERNEL(false, true,  false, false); break;
                    case 3:  CALL_KERNEL(true,  true,  false, false); break;
                    case 4:  CALL_KERNEL(false, false, true,  false); break;
                    case 5:  CALL_KERNEL(true,  false, true,  false); break;
                    case 6:  CALL_KERNEL(false, true,  true,  false); break;
                    case 7:  CALL_KERNEL(true,  true,  true,  false); break;
                    case 8:  CALL_KERNEL(false, false, false, true);  break;
                    case 9:  CALL_KERNEL(true,  false, false, true);  break;
                    case 10: CALL_KERNEL(false, true,  false, true);  break;
                    case 11: CALL_KERNEL(true,  true,  false, true);  break;
                    case 12: CALL_KERNEL(false, false, true,  true);  break;
                    case 13: CALL_KERNEL(true,  false, true,  true);  break;
                    case 14: CALL_KERNEL(false, true,  true,  true);  break;
                    case 15: CALL_KERNEL(true,  true,  true,  true);  break;
                }
                                    


                if(deviceId == 0) {
                    CHECK_CUDA(hipEventRecord(endKernelOnly));
                    CHECK_CUDA(hipEventRecord(startCopyOnly));
                }

                // --- D2H Copy ---
                // Copy the computed batch back to the correct slice of the final host array.
                // The D2H copy will automatically block/ synchronize the kernel for this batch.
                // This could be improved by using streams / hipMemcpyAsync.
                void *d_src_ptr = (isRealOutput ? (void*)d_valueReal_out_batch.get() : (void*)d_valueCmpl_out_batch.get());
                void* h_dest_ptr = (isRealOutput ? (void*)calc->valueReal_out : (void*)calc->valueCmpl_out);

                size_t host_plane_size = (size_t)nPointsZ * nPointsY * nPointsX * number_size;
                size_t device_plane_size = (size_t)deviceParams.nPointsZ_batch * nPointsY * nPointsX * number_size;

                for(int iEig = 0; iEig < calc->nEigOut; ++iEig) {
                    // From: iEig-th slice of GPU batch buffer
                    ptrdiff_t d_offset_bytes = (ptrdiff_t)(iEig * device_plane_size);
                    
                    // To: Global Z-position in the iEig-th slice of host buffer
                    ptrdiff_t h_offset_bytes = (ptrdiff_t)(iEig * host_plane_size + ( (size_t)deviceParams.z_offset_global * nPointsY * nPointsX) * number_size);

                    CHECK_CUDA(hipMemcpy((char*)h_dest_ptr + h_offset_bytes, (char*)d_src_ptr + d_offset_bytes, device_plane_size, hipMemcpyDeviceToHost));
                }


                if(deviceId == 0) {
                    CHECK_CUDA(hipEventRecord(endCopyOnly));
                    CHECK_CUDA(hipEventSynchronize(endCopyOnly));

                    float iterKernel_ms, iterCopy_ms;
                    CHECK_CUDA(hipEventElapsedTime(&iterKernel_ms, startKernelOnly, endKernelOnly));
                    CHECK_CUDA(hipEventElapsedTime(&iterCopy_ms, endKernelOnly, endCopyOnly));

                    totalKernelTime_ms += iterKernel_ms;
                    totalD2HCopyTime_ms += iterCopy_ms;
                }
            }
        }
    } // End of omp parallel region

    // Synchronize all devices
    for(int i = 0; i < numGpus; ++i) {
        CHECK_CUDA(hipSetDevice(i));
        CHECK_CUDA(hipDeviceSynchronize());
    }

    // Switch back to lead to retrieve timing
    CHECK_CUDA(hipSetDevice(0));
    CHECK_CUDA(hipGetLastError());

    // --- Final Timing ---
    CHECK_CUDA(hipEventRecord(endEverything));
    CHECK_CUDA(hipEventSynchronize(endEverything));

    float timeEverything;
    CHECK_CUDA(hipEventElapsedTime(&timeEverything, startEverything, endEverything));


    float overhead = timeEverything - (totalKernelTime_ms + totalD2HCopyTime_ms);
    if(debug)
    printf("\n--- GPU Timing Results ---\n");
    printf("Total Multi-GPU execution time: %.2f ms\n", timeEverything);
    if(debug){
    printf("(Lead) Kernel execution: %.2f ms (%.1f%%)\n", totalKernelTime_ms, (totalKernelTime_ms / timeEverything) * 100.0);
    printf("(Lead) D2H Copy:         %.2f ms (%.1f%%)\n", totalD2HCopyTime_ms, (totalD2HCopyTime_ms / timeEverything) * 100.0);
    }
}
