#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <omp.h>
#include <cstdio>
#include <cmath>
#include <algorithm>
#include <assert.h>

#include "kernel.cuh"
#include "utils.cuh"
#include "slater.cuh"

// more print statements
constexpr bool debug = false; 
// =========================================================================
//  CUDA Kernel.
//  We surely want to separate the arguments into structs for better maintainability.
// =========================================================================
// To avoid branching (dropped at compile time), we template the kernel 4 (3 usable) ways on (isReal, isDensity).
// isPeriodic decides whether to fold coords into unit cell.
// isReal decides whether to use real/complex outArrays, eigenvectors (and adds phases)
// isDensity (requiring isReal) computes single eigenstate density output in valueReal_out of shape (x,y,z,1)
// solely the orbital value is squared, users is responsible for squaring eigenvec beforehand if needed to avoid repeated multiplication. (And multiplying with occupation numbers if needed.)
template <bool isReal, bool isDensity>
__global__ void evaluateKernel(
    const int nPointsX, const int nPointsY, const int nPointsZ_batch, const int z_offset, int nEig, int nOrb, int nStos,
    int maxNPows, int maxNAlphas, int nAtom, int nCell, int nEig_per_pass, const bool isPeriodic,
    const double* __restrict__ origin, const double* __restrict__ gridVecs, const double* __restrict__ eigVecsReal,
    const hipDoubleComplex* __restrict__ eigVecsCmpl, 
    const double* __restrict__ coords, const int* __restrict__ species, const int* __restrict__ iStos,
    const double* __restrict__ latVecs, const double* __restrict__ recVecs2p, const int* __restrict__ kIndexes,
    const hipDoubleComplex* __restrict__ phases,
    const int* __restrict__ sto_angMoms, const int* __restrict__ sto_nPows, const int* __restrict__ sto_nAlphas,
    const double* __restrict__ sto_cutoffsSq, const double* __restrict__ sto_coeffs, const double* __restrict__ sto_alphas,
    double* valueReal_out_batch, hipDoubleComplex* valueCmpl_out_batch)
{
    // isDensity not compatible with isCmplx=!isReal
    assert(!(isDensity && !isReal));
    using AccumT = typename std::conditional<(isReal), double, hipDoubleComplex>::type;
    
    // Each thread gets its own private slice of the shared memory buffer for fast accumulation.
    // We have to chunk the eigenstates into nEig_per_pass due to size constraints.
    // (Cuda doesnt allow templating the shared memory type, so we simply recast it.)
    extern __shared__ double shared_workspace[];
    size_t doubles_per_thread = isReal ? nEig_per_pass : nEig_per_pass * 2;
    AccumT* point_results_pass = reinterpret_cast<AccumT*>(&shared_workspace[threadIdx.x * doubles_per_thread]);


    // --- Thread to point mapping ---
    // Map each thread to unique 1d index
    int idx_in_batch = blockIdx.x * blockDim.x + threadIdx.x;
    int total_points_in_batch = nPointsX * nPointsY * nPointsZ_batch;
    if (idx_in_batch >= total_points_in_batch) return;

    // Map 1d index to point in grid
    int i1 = idx_in_batch % nPointsX;
    int i2 = (idx_in_batch / nPointsX) % nPointsY;
    int i3_batch = idx_in_batch / (nPointsX * nPointsY);
    int i3_global = i3_batch + z_offset;

    // Map point to global coordinates.
    double xyz[3];
    for (int i = 0; i < 3; ++i) {
        xyz[i] = origin[i] + i1 * gridVecs[IDX2F(i, 0, 3)]
                           + i2 * gridVecs[IDX2F(i, 1, 3)]
                           + i3_global * gridVecs[IDX2F(i, 2, 3)];
    }
    // If periodic, fold into cell by discarding the non-fractional part in lattice vector multiples.
    if (isPeriodic) {
        foldCoordsIntoCell(xyz, reinterpret_cast<const double (*)[3]>(latVecs), reinterpret_cast<const double (*)[3]>(recVecs2p));
    }


    // --- Loop over eigenstates in chunks that fit in shared memory ---
    for (int eig_base = 0; eig_base < nEig; eig_base += nEig_per_pass) {
        
        // Initialize the small, per-pass buffer for this thread
        for (int i = 0; i < nEig_per_pass; ++i) {
            if constexpr (isReal || isDensity) {
                point_results_pass[i] = 0.0; 
            } else {
                point_results_pass[i] = make_hipDoubleComplex(0.0, 0.0);
            }
        }

        // The spatial calculation is repeated for each chunk of eigenstates.
        // This is to keep the accumulation in fast shared memory.
        for (int iCell = 0; iCell < nCell; ++iCell) {
            int orbital_idx_counter = 0; 
            for (int iAtom = 0; iAtom < nAtom; ++iAtom) {
                int iSpecies = species[iAtom] - 1;
                double diff[3];
                for (int i = 0; i < 3; ++i) {
                    diff[i] = xyz[i] - coords[IDX3F(i, iAtom, iCell, 3, nAtom)];
                }
                double rr = diff[0] * diff[0] + diff[1] * diff[1] + diff[2] * diff[2];

                for (int iOrb = iStos[iSpecies] - 1; iOrb < iStos[iSpecies + 1] - 1; ++iOrb) {
                    int iL = sto_angMoms[iOrb];
                    if (rr > sto_cutoffsSq[iOrb]) {
                        orbital_idx_counter += 2 * iL + 1;
                        continue;
                    }
                    double r = sqrt(rr);
                    
                    double radialVal = getRadialValue(
                        r, iL, iOrb, sto_nPows[iOrb], sto_nAlphas[iOrb],
                        sto_coeffs, sto_alphas, maxNPows, maxNAlphas);


                    // precompute inverse used across several realTessY calls
                    double inv_r = (r < 1.e-12) ? 0.0 : 1.0 / r;
                    double inv_r2 = inv_r * inv_r;

                    for (int iM = -iL; iM <= iL; ++iM) {
                        double val = radialVal * realTessY(iL, iM, diff, inv_r, inv_r2);
                        if constexpr (isDensity) val *= val; 
                        
                        // Accumulate into the small shared memory buffer for the current chunk
                        for (int iEig_offset = 0; iEig_offset < nEig_per_pass; ++iEig_offset) {
                            int iEig = eig_base + iEig_offset;
                            if (iEig >= nEig) break; // Don't go past the end on the last chunk
                            size_t eig_idx = IDX2F(orbital_idx_counter, iEig, nOrb);
                            if constexpr (isDensity) {
                                point_results_pass[0] += val * eigVecsReal[eig_idx];
                            } else if constexpr (isReal) {
                                point_results_pass[iEig_offset] += val * eigVecsReal[eig_idx];
                            } else {
                                hipDoubleComplex phase = phases[IDX2F(iCell, iEig, nCell)];
                                hipDoubleComplex ev = eigVecsCmpl[eig_idx];
                                hipDoubleComplex psi = hipCmul(make_hipDoubleComplex(val, 0.0), hipCmul(phase, ev));
                                point_results_pass[iEig_offset] = hipCadd(point_results_pass[iEig_offset], psi);
                            }
                        }
                        orbital_idx_counter++;
                    }
                }
            }
        }

        // Write the complete nEig_per_pass chunk to global memory.
        for (int iEig_offset = 0; iEig_offset < nEig_per_pass; ++iEig_offset) {
            int iEig = eig_base + iEig_offset;
            if (iEig >= nEig) break;
            size_t out_idx = IDX4F(i1, i2, i3_batch, iEig, nPointsX, nPointsY, nPointsZ_batch);

            if constexpr (isReal) {
                valueReal_out_batch[out_idx] = point_results_pass[iEig_offset];
                if constexpr (isDensity) break; 
            
            } else {
                valueCmpl_out_batch[out_idx] = point_results_pass[iEig_offset];
            }
        }
    }
}



// =========================================================================
//  C++ Host Interface (callable from C/Fortran)
// =========================================================================
extern "C" void evaluate_on_device_c(
    const int nPointsX, const int nPointsY, const int nPointsZ,
    const int nEig, const int nEigOut, const int nOrb, const int nStos,
    const int maxNPows, const int maxNAlphas,
    const int nAtom, const int nCell, const int nSpecies,
    const int isReal, const int isPeriodic, const int isDensityCalc,
    const double* h_origin,
    const double* h_gridVecs,
    const double* h_eigVecsReal,
    const hipDoubleComplex* h_eigVecsCmpl,
    const double* h_coords,
    const int* h_species,
    const int* h_iStos,
    const double* h_latVecs,
    const double* h_recVecs2p,
    const int* h_kIndexes,
    const hipDoubleComplex* h_phases,
    const int* h_sto_angMoms,
    const int* h_sto_nPows,
    const int* h_sto_nAlphas,
    const double* h_sto_cutoffsSq,
    const double* h_sto_coeffs,
    const double* h_sto_alphas,
    double* h_valueReal_out,
    hipDoubleComplex* h_valueCmpl_out
){
    if (nEig == 0 || nPointsZ == 0) return; // Nothing to do
    if (isDensityCalc) {
        assert(isReal);
        assert(nEigOut == 1);
    } else {
        assert(nEigOut == nEig);
    }
    
    
    // We currently assume a hardcoded maximum for the number of powers.
    if (maxNPows > STO_MAX_POWS) {
        fprintf(stderr, "Error: maxNPows (%d) exceeds STO_MAX_POWS (%d)\n", maxNPows, STO_MAX_POWS);
        exit(EXIT_FAILURE);
    }

    size_t total_size_valueReal = (size_t)nPointsX * nPointsY * nPointsZ * nEig * sizeof(double);
    // Timing events.
    hipEvent_t startEverything, endEverything;
    hipEvent_t startKernelOnly, endKernelOnly, startCopyOnly, endCopyOnly;
    float totalKernelTime_ms = 0.0f;
    float totalD2HCopyTime_ms = 0.0f;
    CHECK_CUDA(hipEventCreate(&startEverything));
    CHECK_CUDA(hipEventCreate(&endEverything));
    CHECK_CUDA(hipEventCreate(&startKernelOnly));
    CHECK_CUDA(hipEventCreate(&endKernelOnly));
    CHECK_CUDA(hipEventCreate(&startCopyOnly));
    CHECK_CUDA(hipEventCreate(&endCopyOnly));
    CHECK_CUDA(hipEventRecord(startEverything));


    // --- Multi-GPU Setup ---
    int numGpus;
    CHECK_CUDA(hipGetDeviceCount(&numGpus));
    if (numGpus == 0) {
        fprintf(stderr, "Error: No CUDA-enabled GPUs found.\n");
        exit(EXIT_FAILURE);
    }
    printf("Found %d GPUs.", numGpus);

#ifndef _OPENMP
    if (numGpus > 1) {
    printf("\nWARNING: Code not compiled with OpenMP support (-fopenmp). Falling back to single-GPU mode.\n");
    numGpus = 1;
    printf("Running on GPU 0 only.\n");
    }
#endif

    // Use OMP to split across available GPUs
    #pragma omp parallel num_threads(numGpus) 
    {
        int deviceId = omp_get_thread_num();
        CHECK_CUDA(hipSetDevice(deviceId));

        // --- Work Distribution: Divide Z-slices among GPUs ---
        int z_slices_per_gpu = nPointsZ / numGpus;
        int z_start_for_device = deviceId * z_slices_per_gpu;
        int z_count_for_device = (deviceId == numGpus - 1) ? (nPointsZ - z_start_for_device) : z_slices_per_gpu;

        if (z_count_for_device > 0) {
            // --- Per-GPU Data Allocation and H2D Copy ---
            // Each thread allocates data on its own assigned GPU.
            DeviceBuffer<double> d_origin(h_origin, 3);
            DeviceBuffer<double> d_gridVecs(h_gridVecs, 9); 
            DeviceBuffer<double> d_coords(h_coords, (size_t)3 * nAtom * nCell);
            DeviceBuffer<int>    d_species(h_species, nAtom);
            DeviceBuffer<int>    d_iStos(h_iStos, nSpecies + 1);
            DeviceBuffer<int>    d_sto_angMoms(h_sto_angMoms, nStos);
            DeviceBuffer<int>    d_sto_nPows(h_sto_nPows, nStos);
            DeviceBuffer<int>    d_sto_nAlphas(h_sto_nAlphas, nStos);
            DeviceBuffer<double> d_sto_cutoffsSq(h_sto_cutoffsSq, nStos);
            DeviceBuffer<double> d_sto_coeffs(h_sto_coeffs, (size_t)maxNPows * maxNAlphas * nStos);
            DeviceBuffer<double> d_sto_alphas(h_sto_alphas, (size_t)maxNAlphas * nStos);
            // Real / complex stuff (half left as nullptr)
            DeviceBuffer<double> d_eigVecsReal;
            DeviceBuffer<hipDoubleComplex> d_eigVecsCmpl;
            DeviceBuffer<hipDoubleComplex> d_phases;
            DeviceBuffer<int> d_kIndexes;
            DeviceBuffer<double> d_latVecs;
            DeviceBuffer<double> d_recVecs2p;
            if (isReal) {
                d_eigVecsReal = DeviceBuffer<double>(h_eigVecsReal, (size_t)nOrb * nEig);
            } else {
                d_eigVecsCmpl = DeviceBuffer<hipDoubleComplex>(h_eigVecsCmpl, (size_t)nOrb * nEig);
                d_phases = DeviceBuffer<hipDoubleComplex>(h_phases, (size_t)nCell * nEig);
                d_kIndexes = DeviceBuffer<int>(h_kIndexes, nEig);
            }
            if (isPeriodic) {
                d_latVecs = DeviceBuffer<double>(h_latVecs, 9);
                d_recVecs2p = DeviceBuffer<double>(h_recVecs2p, 9);
            }

            // --- Per-GPU Kernel Configuration ---
            int block_size = 256;
            hipDeviceProp_t prop;
            CHECK_CUDA(hipGetDeviceProperties(&prop, deviceId));
            
            // Determine available shared memory for nEig_per_pass
            size_t available_shared = prop.sharedMemPerBlock * 0.95;
            size_t number_size = (isReal || isDensityCalc) ? sizeof(double) : sizeof(hipDoubleComplex);
            int nEig_per_pass = available_shared / (block_size * number_size);
            if (nEig_per_pass == 0) nEig_per_pass = 1;
            if (nEig_per_pass > nEig) nEig_per_pass = nEig;
            size_t shared_mem_for_pass = (size_t)nEig_per_pass * block_size * number_size;

            // Density stored as single value per point
            if (isDensityCalc) {
                nEig_per_pass = nEig;
                shared_mem_for_pass = block_size * number_size; 
            }

            
            // Determine the number of Z-slices to process in a single batch
            size_t free_mem, total_mem;
            CHECK_CUDA(hipMemGetInfo(&free_mem, &total_mem));
            size_t available_for_batch = static_cast<size_t>(free_mem * 0.8);
            size_t z_slice_size_bytes = (size_t)nPointsX * nPointsY * nEig * number_size;
            
            // Determine max Z-slices that can fit in available (global) memory
            int z_batch_size = z_count_for_device; 
            if (z_slice_size_bytes > 0 && ((size_t)z_count_for_device * z_slice_size_bytes) > available_for_batch) {
                z_batch_size = available_for_batch / z_slice_size_bytes;
                if (z_batch_size == 0) z_batch_size = 1;
            }

            #pragma omp critical
            if (deviceId == 0 && debug) {
                printf("\n--- GPU %d (Lead) Configuration ---\n", deviceId);
                printf("  Z-slice workload: %d (from index %d to %d)\n", z_count_for_device, z_start_for_device, z_start_for_device + z_count_for_device - 1);
                printf("  Block size: %d threads, %zub shared mem per block, %d eigs per pass\n",
                    block_size, shared_mem_for_pass, nEig_per_pass);
                printf(" (Free device mem: %.2f GB, Grid size: %d x %d x %d (x %d eigs) = %.2f GB)\n",
                    free_mem / 1e9, nPointsX, nPointsY, nPointsZ, nEig,
                    total_size_valueReal / 1e9);
                printf("  Processing Z-slices in batches of %d\n", z_batch_size);

            }

            // Per-GPU batch buffer for the output
            const int nEig_out = (isDensityCalc) ? 1 : nEig; // Density output is always 1 eig
            size_t batch_buffer_size_elems = (size_t)nPointsX * nPointsY * std::min(z_count_for_device, z_batch_size) * nEig_out;
            DeviceBuffer<hipDoubleComplex> d_valueCmpl_out_batch;
            DeviceBuffer<double> d_valueReal_out_batch;
            if (isReal || isDensityCalc) {
                d_valueReal_out_batch = DeviceBuffer<double>(batch_buffer_size_elems);
            } else {
                d_valueCmpl_out_batch = DeviceBuffer<hipDoubleComplex>(batch_buffer_size_elems);
            }


            // --- Per-GPU Kernel Execution Loop ---
            // This loop iterates over the Z-slices assigned to *this* GPU.
            for (int z_offset_in_device_chunk = 0; z_offset_in_device_chunk < z_count_for_device; z_offset_in_device_chunk += z_batch_size) {
                int current_nPointsZ_batch = std::min(z_batch_size, z_count_for_device - z_offset_in_device_chunk);
                int total_points_in_batch = nPointsX * nPointsY * current_nPointsZ_batch;
                if (total_points_in_batch == 0) continue;

                // The global z_offset is what the kernel needs to calculate correct coordinates
                int z_offset_global = z_start_for_device + z_offset_in_device_chunk;
                int grid_size = (total_points_in_batch + block_size - 1) / block_size;
                if(deviceId == 0) {
                    CHECK_CUDA(hipEventRecord(startKernelOnly));
                }

                #define LAUNCH_KERNEL(R, D) \
                evaluateKernel<R, D><<<grid_size, block_size, shared_mem_for_pass>>>( \
                    nPointsX, nPointsY, current_nPointsZ_batch, z_offset_global, nEig, nOrb, nStos, \
                    maxNPows, maxNAlphas, nAtom, nCell, nEig_per_pass, isPeriodic, \
                    d_origin.get(), d_gridVecs.get(), d_eigVecsReal.get(), d_eigVecsCmpl.get(), \
                    d_coords.get(), d_species.get(), d_iStos.get(), \
                    d_latVecs.get(), d_recVecs2p.get(), d_kIndexes.get(), d_phases.get(), \
                    d_sto_angMoms.get(), d_sto_nPows.get(), d_sto_nAlphas.get(), \
                    d_sto_cutoffsSq.get(), d_sto_coeffs.get(), d_sto_alphas.get(), \
                    d_valueReal_out_batch.get(), d_valueCmpl_out_batch.get()) 

                if (isReal) {
                    if (isDensityCalc) {
                        LAUNCH_KERNEL(true, true);
                    } else {
                        LAUNCH_KERNEL(true, false);
                    }
                } else {
                    LAUNCH_KERNEL(false, false);
                }
                
                #undef LAUNCH_KERNEL

                if(deviceId == 0) {
                    CHECK_CUDA(hipEventRecord(endKernelOnly));
                    CHECK_CUDA(hipEventRecord(startCopyOnly));
                }

                // --- Per-GPU D2H Copy ---
                // Copy the computed batch back to the correct slice of the final host array.
                // The D2H copy will automatically block/ synchronize the kernel for this batch.
                for(int iEig = 0; iEig < nEig_out; ++iEig) {
                    size_t plane_size_bytes = (size_t)current_nPointsZ_batch * nPointsY * nPointsX * number_size;

                    // Source pointer in this GPU's batch buffer
                    double* d_src_ptr_eig = d_valueReal_out_batch.get();
                    double* h_dest_ptr_eig = h_valueReal_out;
                    bool isRealOutput = (isReal || isDensityCalc);
                    if (!isRealOutput) {
                        d_src_ptr_eig = reinterpret_cast<double*>(d_valueCmpl_out_batch.get());
                        h_dest_ptr_eig = reinterpret_cast<double*>(h_valueCmpl_out);

                    }
                    d_src_ptr_eig +=  (size_t)iEig * current_nPointsZ_batch * nPointsY * nPointsX;
                    
                    // Destination pointer in the final large host output array.
                    // The offset is calculated using the GLOBAL Z-offset.
                    h_dest_ptr_eig += (size_t)iEig * nPointsZ * nPointsY * nPointsX + (size_t)z_offset_global * nPointsY * nPointsX;
                    CHECK_CUDA(hipMemcpy(h_dest_ptr_eig, d_src_ptr_eig, plane_size_bytes, hipMemcpyDeviceToHost));
                }
                if(deviceId == 0) {
                    CHECK_CUDA(hipEventRecord(endCopyOnly));
                    CHECK_CUDA(hipEventSynchronize(endCopyOnly));

                    float iterKernel_ms, iterCopy_ms;
                    CHECK_CUDA(hipEventElapsedTime(&iterKernel_ms, startKernelOnly, endKernelOnly));
                    CHECK_CUDA(hipEventElapsedTime(&iterCopy_ms, endKernelOnly, endCopyOnly));

                    totalKernelTime_ms += iterKernel_ms;
                    totalD2HCopyTime_ms += iterCopy_ms;
                }
            }
        }
    } // End of omp parallel region

    // Synchronize all devices
    for(int i = 0; i < numGpus; ++i) {
        CHECK_CUDA(hipSetDevice(i));
        CHECK_CUDA(hipDeviceSynchronize());
    }

    // Switch back to lead to retrieve timing
    CHECK_CUDA(hipSetDevice(0));
    CHECK_CUDA(hipGetLastError());

    // --- Final Timing ---
    CHECK_CUDA(hipEventRecord(endEverything));
    CHECK_CUDA(hipEventSynchronize(endEverything));

    float timeEverything;
    CHECK_CUDA(hipEventElapsedTime(&timeEverything, startEverything, endEverything));


    float overhead = timeEverything - (totalKernelTime_ms + totalD2HCopyTime_ms);
    if(debug)
    printf("\n--- GPU Timing Results ---\n");
    printf("Total Multi-GPU execution time: %.2f ms\n", timeEverything);
    if(debug){
    printf("(Lead) Kernel execution: %.2f ms (%.1f%%)\n", totalKernelTime_ms, (totalKernelTime_ms / timeEverything) * 100.0);
    printf("(Lead) D2H Copy:         %.2f ms (%.1f%%)\n", totalD2HCopyTime_ms, (totalD2HCopyTime_ms / timeEverything) * 100.0);
    }
}
